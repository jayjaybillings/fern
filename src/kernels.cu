#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernels.cuh"

extern __shared__ char dsmem[];
__device__ fern_real *scratch_space;

__global__ void integrateNetwork(
	Network network,
	IntegrationData integrationData,
	Globals *globalsPtr
)
{
	const int tid = threadIdx.x;
	
	Globals &globals = *globalsPtr;

	/*
	   TEMPORARY
	   Variables are declared with local pointers.
	   This is to ease refactoring and allow easy
	   maneuvering with dynamic shared memory.
	*/

	/* Declare local pointers for Globals arrays. */

	fern_real *Flux;
	fern_real *Fplus;
	fern_real *Fminus;
	fern_real *Rate;
	fern_real *massNum;
	fern_real *X;
	fern_real *Fdiff;
	fern_real *Yzero;
	fern_real *FplusSum;
	fern_real *FminusSum;
	
	/* Declare local variables for Network struct. */

	const unsigned short numberSpecies = network.species;
	const unsigned short numberReactions = network.reactions;
	const unsigned short totalFplus = network.totalFplus;
	const unsigned short totalFminus = network.totalFminus;

	unsigned char *Z;
	unsigned char *N;

	fern_real *FplusFac;
	fern_real *FminusFac;

	unsigned short *MapFplus;
	unsigned short *MapFminus;

	unsigned short *FplusMax;
	unsigned short *FminusMax;

	const fern_real massTol = network.massTol;
	const fern_real fluxFrac = network.fluxFrac;

	/* Declare pointer variables for Partial Equilibrium arrays */
    int *ReacParent = new int [numberReactions];
	/* Declare pointer variables for IntegrationData arrays.  */

	fern_real *Y;

	/* Assign globals pointers. */
	
	Flux = globals.Flux;
	Fplus = globals.Fplus;
	Fminus = globals.Fminus;
	Rate = globals.rate;
	massNum = globals.massNum;
	X = globals.X;
	Fdiff = globals.Fdiff;
	Yzero = globals.Yzero;
	FplusSum = globals.FplusSum;
	FminusSum = globals.FminusSum;

	/* Assign Network pointers. */

	Z = network.Z;
	N = network.N;
	FplusFac = network.FplusFac;
	FminusFac = network.FminusFac;
	MapFplus = network.MapFplus;
	MapFminus = network.MapFminus;
	FplusMax = network.FplusMax;
	FminusMax = network.FminusMax;

	/* Assign Partial Equilibrium pointers. */

	/* Assign IntegrationData pointers. */
	
	Y = integrationData.Y;

	/*
	   TODO
	   Evaluate shared bank conflicts to avoid serializing the shared
	   memory accesses.

	   Ref: https://developer.nvidia.com/content/using-shared-memory-cuda-cc
	*/

	/* Allocate shared memory. */

	int shared_pos;

	shared_pos = 0;

	FplusSum = (fern_real *) (dsmem + shared_pos);
	shared_pos += network.species * sizeof(fern_real);
	FminusSum = (fern_real *) (dsmem + shared_pos);
	shared_pos += network.species * sizeof(fern_real);

	/*
	   Allocate dsmem scratch space (see NDreduceSum).
	   To be safe, ensure numberReactions * sizeof(fern_real)
	   bytes are available, although this can be trimmed
	   quite a bit for production.
	*/

	scratch_space = (fern_real *) (dsmem + shared_pos);
	shared_pos += network.reactions * sizeof(fern_real);

	if (tid == 0) printf("%d bytes of dsmem used.\n", shared_pos);

	__syncthreads();


	/* Static shared memory */
	
	__shared__ fern_real maxFlux;
	__shared__ fern_real sumX;
	__shared__ fern_real t;
	__shared__ fern_real dt;
	__shared__ unsigned int timesteps;
	
	fern_real sumXLast;

	/* Compute the preFac vector. */
	
	if (tid == 0)
	{
		for (int i = 0; i < network.reactions; i++)
		{
			#ifdef FERN_SINGLE
				globals.preFac[i] = network.statFac[i] *
					powf(integrationData.rho, network.numReactingSpecies[i] - 1);
			#else
				globals.preFac[i] = network.statFac[i] *
					pow(integrationData.rho, network.numReactingSpecies[i] - 1);
			#endif
		}
	}

	/* Compute the rate values. */

	if (tid == 0)
	{
		/*
		   Compute the temperature-dependent factors for the rates.
		   Since we assume the GPU integration to be done at constant
		   temperature and density, these only need be calculated once
		   per GPU call.
		*/
		
		fern_real T93 = cbrt(integrationData.T9);
		fern_real t1 = 1 / integrationData.T9;
		fern_real t2 = 1 / T93;
		fern_real t3 = T93;
		fern_real t4 = integrationData.T9;
		fern_real t5 = T93 * T93 * T93 * T93 * T93;
		fern_real t6 = log(integrationData.T9);
		
		for (int i = 0; i < network.reactions; i++)
		{
			#ifdef FERN_SINGLE
				Rate[i] = globals.preFac[i] * expf(
					     network.P[0][i] + t1 * network.P[1][i] +
					t2 * network.P[2][i] + t3 * network.P[3][i] +
					t4 * network.P[4][i] + t5 * network.P[5][i] +
					t6 * network.P[6][i]);
			#else
	 			Rate[i] = globals.preFac[i] * exp(
					     network.P[0][i] + t1 * network.P[1][i] +
					t2 * network.P[2][i] + t3 * network.P[3][i] +
					t4 * network.P[4][i] + t5 * network.P[5][i] +
					t6 * network.P[6][i]);
			#endif

		}
	}

	/* Author: Daniel Shyles */
	/* Begin Partial Equilibrium calculation */
    int RGParent;
	const bool displayRGdata = true;
    fern_real y_a;
    fern_real y_b;
    fern_real y_c;
    fern_real y_d;
    fern_real y_e;
	//constants for coefficient and relative abundance calculations
	fern_real c1;
	fern_real c2;
	fern_real c3;
	fern_real c4;
	//TODO: Add this back in -->	fern_real c4 = y_a+y_e; 
	//coefficients for q and theoretical equilibrium abundance calculation
	fern_real a;
	fern_real b;
	fern_real c;
	//to be used in RG Class 5 PE calculation;
	fern_real alpha;
	fern_real beta;
	fern_real gamma;
	fern_real q;
	fern_real y_eq_a; 
	fern_real y_eq_b; 
	fern_real y_eq_c; 
	fern_real y_eq_d; 
	fern_real y_eq_e; 
	//set tolerance for determining partial equilibrium
	fern_real tolerance = .01;
	//result of PE determination... if PE_val < tolerance, we're in equilibrium
	fern_real PE_val_a;
	fern_real PE_val_b;
	fern_real PE_val_c;
	fern_real PE_val_d;
	fern_real PE_val_e;
	//if in equilibrium I'll set this to 1
	int eq;

	// Population at time t (unnecessary for PE calculation as we have current populations):
	//fern_real y_0;
    //fern_real phi = ((2*a*y_0)+b+sqrt(-q))/((2*a*y_0)+b-sqrt(-q));
	//fern_real y_it = (-.5/a)*(b+(sqrt(-q)*((1+phi*expf(-sqrt(-q)*t))/(1-phi*expf(-sqrt(-q)*t)))));

	if (tid == 0) {
		//first loop through reactions to set up RG parents
		int numRG = 0;
		for(int i = 0; i < network.reactions; i++) {
		    if(network.ReacGroups[i] != 0) {
				numRG++;
			    //This indicates a new reaction group
				RGParent = i;
		    }
	        ReacParent[i] = RGParent;
		}
	    fern_real kf;
		fern_real kr;
	    fern_real *final_k[2];
		//set up array of final reaction rates for each RG
		//doing it this way saves memory, as we don't need two arrays with size numberReactions, only numRG
		for(int m = 0; m < 2; m++)
			final_k[m] = new fern_real [numRG];
		if(displayRGdata)
			printf("Start Reaction Group Data\nNumber Reaction Groups: %d\n\n",numRG);
		//second to calculate final reaciton rates for each RG
		for(int i = 0; i < network.reactions; i++) {

			if(displayRGdata && network.ReacGroups[i] != 0) {
				printf("RG Class: %d\nRG ID (Parent): %d\n", network.ReacGroups[i], i);
				//output numReactingSpecies and numProducts for Parent of RG
                printf("numReacting: %d, numProducts: %d\n", network.numReactingSpecies[i], network.PEnumProducts[i]);			
				if(network.ReacGroups[i] == 1) {		
					printf("Reactant SID: %d; Product SID: %d\n",network.reactant[0][i], network.product[0][i]);
				} 
				else if(network.ReacGroups[i] == 2) {
                    printf("Reactant SID: %d, %d; Product SID: %d\n",network.reactant[0][i], network.reactant[1][i], network.product[0][i]);
                }
                else if(network.ReacGroups[i] == 3) {
                    printf("Reactant SID: %d, %d, %d; Product SID: %d\n",network.reactant[0][i], network.reactant[1][i], network.reactant[2][i], network.product[0][i]);
                }
                else if(network.ReacGroups[i] == 4) {
                    printf("Reactant SID: %d, %d; Product SID: %d, %d\n",network.reactant[0][i], network.reactant[1][i], network.product[0][i], network.product[1][i]);
                }
                else if(network.ReacGroups[i] == 5) {
                    printf("Reactant SID: %d, %d; Product SID: %d, %d, %d\n",network.reactant[0][i], network.reactant[1][i], network.product[0][i], network.product[1][i], network.product[2][i]);
                }
				printf("-----\n|\n");
			}				
			if(displayRGdata)
				printf("Reaction ID: %d\nRG Member ID: %d\nForward Reaction (+-Q): %d\nRate: %f\n|\n", i, network.RGmemberIndex[i], network.pnQ[i], Rate[i]);

            //if RGmemberindex is greater (or equal for RGmemberindex[i] = RGmemberindex[i+1] = 0 than next one, then end of Reaction Group
            if(network.RGmemberIndex[i] >= network.RGmemberIndex[i+1]) {
                //get forward and reverse rates for all reactions within group, starting with i-network.RGmemberIndex[i], and ending with i.
                kf = 0; //forward rate
                kr = 0; //reverse rate
                //iterate through each RGmember and calculate the total rate from forward and reverse reactions
                for(int n = network.RGmemberIndex[i]; n >= 0; n--) {
                    //add the rate to forward reaction
                    /****************************************************************************************************************************
                     *TODO rework this such that reactions with multiple sets of parameters (n > 1, three or more reactions per reaction group) *
                     *have their final rates (kf, kr) calculated properly. Currently being summed, but some other operation is necessary here...*
                     ****************************************************************************************************************************/
                    if(network.pnQ[i-n] == 1) {
                        kf += Rate[i-n];
                    } else {
                    //add the rate to reverse reaction
                        kr += Rate[i-n];
                    }
                    //printf("current forward Rate for RG with Parent %d = %f and kf = %f \n ", ReacParent[i], Rate[i-n], kf);
                    //printf("current reverse Rate for RG with Parent %d = %f and kr = %f \n ", ReacParent[i], Rate[i-n], kr);
                }
                final_k[0][ReacParent[i]] = kf;
                final_k[1][ReacParent[i]] = kr;
                if(displayRGdata) {
                    printf("-----\n");
                    printf("Final Forward Rate: kf = %f \n", final_k[0][ReacParent[i]]);
                    printf("Final Reverse Rate: kr = %f \n", final_k[1][ReacParent[i]]);
                    printf("\n\n\n");
                }
            }
		}

		//final partial equilibrium loop for calculating equilibrium
		const bool displayPEdata = true;
		if(displayPEdata)
			printf("Start Partial Equilibrium Data\n");		
		for(int i = 0; i < numberReactions; i++) {
	        //reset RG reactant and product populations
            y_a = 0;
            y_b = 0;
            y_c = 0;
            y_d = 0;
            y_e = 0;
			if(network.ReacGroups[i] !=0) {
				//Get current population for each reactant and product of this RG
				//TODO: figure out how to differentiate between a neutron as reactant/product and a null entry, as n has Isotope species ID = 0.
				//TODO: Something to watch out for: if a reaction has, for example, three reactants and two products such as in RGclass 5,
				// will it be presented first (RGParent) as a+b+c --> d+e, or might the RGParent have the reverse set up, a+b --> c+d+e
				// if the latter occurs, we'll need to add some logic to account for that. Right now, assuming that all RGParents are set up
				// in the former scenario. This would then be another instance where we'll need to differentiate between neutrons and null 
				// in the reactant and product arrays.
				if(network.ReacGroups[i] == 1) {
					y_a = Y[network.reactant[0][i]];
					y_b = Y[network.product[0][i]];
                   //set specific constraints and coefficients for RGclass 2
                    c1 = y_a+y_b;
                    a = 0;
                    b = -final_k[0][i];
                    c = final_k[1][i];
                    q = 0;

                    //theoretical equilibrium population of given species
                    y_eq_a = -c/b;
                    y_eq_b = c1-y_eq_a;

                    //is each reactant and product in equilibrium?
                    PE_val_a = abs(y_a-y_eq_a)/(y_eq_a);
                    PE_val_b = abs(y_b-y_eq_b)/(y_eq_b);
                    if(PE_val_a < tolerance && PE_val_b < tolerance) {
                        eq = 1;
                    } else {
                        eq = 0;
                    }

                    if(displayPEdata && !isnan(PE_val_a) && !isnan(PE_val_b)) {
                        //only print if there are populations with which to calculate
                        printf("Reaction Group ID: %d, RG Class: %d\n",i, network.ReacGroups[i]);
                        for (int n = 0; n < 2; n++)
                            printf("Reactant[%d]: %d\n",n, network.reactant[n][i]);
                        for (int n = 0; n < 2; n++)
                            printf("Product[%d]: %d\n",n, network.product[n][i]);

                        printf("PE_val for y_a (Species: %d): %f\nPE_val for y_b (Species: %d): %f\nIs this RG in equilibrium?: %d\n",network.reactant[0][i],PE_val_a,network.product[0][i],PE_val_b,eq);
                        printf("\n\n");
                    }
				} 
				else if(network.ReacGroups[i] == 2) {
                    y_a = Y[network.reactant[0][i]];
                    y_b = Y[network.reactant[1][i]];
                    y_c = Y[network.product[0][i]];
                    //set specific constraints and coefficients for RGclass 2
                    c1 = y_b-y_a;
                    c2 = y_b+y_c;
                    a = -final_k[0][i];
                    b = -(c1*final_k[0][i]+final_k[1][i]);
                    c = final_k[1][i]*(c2-c1);
                    q = (4*a*c)-(b*b);

                    //theoretical equilibrium population of given species
                    y_eq_a = ((-.5/a)*(b+sqrt(-q)));
                    y_eq_b = y_eq_a+c1;
                    y_eq_c = c2-y_eq_b;

                    //is each reactant and product in equilibrium?
                    PE_val_a = abs(y_a-y_eq_a)/(y_eq_a);
                    PE_val_b = abs(y_b-y_eq_b)/(y_eq_b);
                    PE_val_c = abs(y_c-y_eq_c)/(y_eq_c);
                    if(PE_val_a < tolerance && PE_val_b < tolerance && PE_val_c < tolerance) {
                        eq = 1;
                    } else {
                        eq = 0;
                    }

                    if(displayPEdata && !isnan(PE_val_a) && !isnan(PE_val_b) && !isnan(PE_val_c)) {
                        //only print if there are populations with which to calculate
                        printf("Reaction Group ID: %d, RG Class: %d\n",i, network.ReacGroups[i]);
                        for (int n = 0; n < 2; n++)
                            printf("Reactant[%d]: %d\n",n, network.reactant[n][i]);
                        for (int n = 0; n < 2; n++)
                            printf("Product[%d]: %d\n",n, network.product[n][i]);

						printf("PE_val for y_a (Species: %d): %f\nPE_val for y_b (Species: %d): %f\nPE_val for y_c (Species: %d): %f\nIs this RG in equilibrium?: %d\n",
							network.reactant[0][i],PE_val_a,
							network.reactant[1][i],PE_val_b,
							network.product[1][i],PE_val_c,
							eq
						);
                        printf("\n\n");
                    }
                }
                else if(network.ReacGroups[i] == 3) {
                    y_a = Y[network.reactant[0][i]];
                    y_b = Y[network.reactant[1][i]];
                    y_c = Y[network.reactant[2][i]];
                    y_d = Y[network.product[0][i]];
                    //set specific constraints and coefficients for RGclass 3
                    c1 = y_a-y_b;
                    c2 = y_a-y_c;
                    c3 = ((1/3)*(y_a+y_b+y_c))+y_d;
                    a = final_k[0][i]*(c1+c2)-final_k[0][i]*y_a;
                    b = -((final_k[0][i]*c1*c2)+final_k[1][i]);
                    c = final_k[1][i]*(c3+(c1/3)+(c2/3));
                    q = (4*a*c)-(b*b);

                    //theoretical equilibrium population
                    y_eq_a = ((-.5/a)*(b+sqrt(-q)));
                    y_eq_b = y_eq_a-c1;
                    y_eq_c = y_eq_a-c2;
                    y_eq_d = c3-y_eq_a+((1/3)*(c1+c2));

                    //is each reactant and product in equilibrium?
                    PE_val_a = abs(y_a-y_eq_a)/(y_eq_a);
                    PE_val_b = abs(y_b-y_eq_b)/(y_eq_b);
                    PE_val_c = abs(y_c-y_eq_c)/(y_eq_c);
                    PE_val_d = abs(y_d-y_eq_d)/(y_eq_d);
                    if(PE_val_a < tolerance && PE_val_b < tolerance && PE_val_c < tolerance && PE_val_d < tolerance) {
                        eq = 1;
                    } else {
                        eq = 0;
                    }

                    if(displayPEdata && !isnan(PE_val_a) && !isnan(PE_val_b) && !isnan(PE_val_c) && !isnan(PE_val_d)) {
                        //only print if there are populations with which to calculate
                        printf("Reaction Group ID: %d, RG Class: %d\n",i, network.ReacGroups[i]);
                        for (int n = 0; n < 2; n++)
                            printf("Reactant[%d]: %d\n",n, network.reactant[n][i]);
                        for (int n = 0; n < 2; n++)
                            printf("Product[%d]: %d\n",n, network.product[n][i]);

                        printf("PE_val for y_a (Species: %d): %f\nPE_val for y_b (Species: %d): %f\nPE_val for y_c (Species: %d): %f\nPE_val for y_d (Species: %d): %f\nIs this RG in equilibrium?: %d\n",
                            network.reactant[0][i],PE_val_a,
                            network.reactant[1][i],PE_val_b,
                            network.reactant[2][i],PE_val_c,
                            network.product[0][i],PE_val_d,
                            eq
                        );
                        printf("\n\n");
                    }

                }
                else if(network.ReacGroups[i] == 4) {
                    y_a = Y[network.reactant[0][i]];
                    y_b = Y[network.reactant[1][i]];
                    y_c = Y[network.product[0][i]];
                    y_d = Y[network.product[1][i]];

                    //set specific constraints and coefficients for RGclass 4
                    c1 = y_a-y_b;
                    c2 = y_a+y_c;
                    c3 = y_a+y_d;
                    a = final_k[1][i]-final_k[0][i];
                    b = -(final_k[1][i]*(c2+c3))+(final_k[0][i]*c1);
                    c = final_k[1][i]*c2*c3;
					q = (4*a*c)-(b*b);

					//calculate theoretical equilibrium value
					y_eq_a = ((-.5/a)*(b+sqrt(-q)));	
					y_eq_b = y_eq_a-c1;
					y_eq_c = c2-y_eq_a;
					y_eq_d = c3-y_eq_a;
				
					//is each reactant and product in equilibrium?	
					PE_val_a = abs(y_a-y_eq_a)/(y_eq_a);
					PE_val_b = abs(y_b-y_eq_b)/(y_eq_b);
					PE_val_c = abs(y_c-y_eq_c)/(y_eq_c);
					PE_val_d = abs(y_d-y_eq_d)/(y_eq_d);
					if(PE_val_a < tolerance && PE_val_b < tolerance && PE_val_c < tolerance && PE_val_d < tolerance) {
						eq = 1;
					} else {
						eq = 0;
					}

					if(displayPEdata && !isnan(PE_val_a) && !isnan(PE_val_b) && !isnan(PE_val_c) && !isnan(PE_val_d)) {
                        //only print if there are populations with which to calculate
						printf("Reaction Group ID: %d, RG Class: %d\n",i, network.ReacGroups[i]);
						for (int n = 0; n < 2; n++)
							printf("Reactant[%d]: %d\n",n, network.reactant[n][i]);
						for (int n = 0; n < 2; n++)
							printf("Product[%d]: %d\n",n, network.product[n][i]);

                        printf("PE_val for y_a (Species: %d): %f\nPE_val for y_b (Species: %d): %f\nPE_val for y_c (Species: %d): %f\nPE_val for y_d (Species: %d): %f\nIs this RG in equilibrium?: %d\n",
                            network.reactant[0][i],PE_val_a,
                            network.reactant[1][i],PE_val_b,
                            network.product[0][i],PE_val_c,
                            network.product[1][i],PE_val_d,
                            eq
                        );
						printf("\n\n");
					}
				}
				else if(network.ReacGroups[i] == 5) {
                    y_a = Y[network.reactant[0][i]];
                    y_b = Y[network.reactant[1][i]];
                    y_c = Y[network.product[0][i]];
                    y_d = Y[network.product[1][i]];
					y_e = Y[network.product[2][i]];

                    //set specific constraints and coefficients for RGclass 5
                    c1 = y_a+(1/3)*(y_c+y_d+y_e);
                    c2 = y_a-y_b;
                    c3 = y_c-y_d;
                    c4 = y_c-y_e;
                    a = (((3*c1)-y_a)*final_k[1][i])-final_k[0][i];
					alpha = c1+((1/3)*(c3+c4));	
					beta = c1-(2*c3/3)+(c4/3);	
					gamma = c1+(c3/3)-(2*c4/3);	
                    b = -(c2*final_k[0][i])-(((alpha*beta)+(alpha*gamma)+(beta*gamma))*final_k[1][i]);
                    c = final_k[1][i]*alpha*beta*gamma;
                    q = (4*a*c)-(b*b);

					//calculate theoretical equilibrium values
                    y_eq_a = ((-.5/a)*(b+sqrt(-q)));
                    y_eq_b = y_eq_a-c2;
                    y_eq_c = alpha-y_eq_a;
                    y_eq_d = beta-y_eq_a;
                    y_eq_e = gamma-y_eq_a;

                    //is each reactant and product in equilibrium?
                    PE_val_a = abs(y_a-y_eq_a)/(y_eq_a);
                    PE_val_b = abs(y_b-y_eq_b)/(y_eq_b);
                    PE_val_c = abs(y_c-y_eq_c)/(y_eq_c);
                    PE_val_d = abs(y_d-y_eq_d)/(y_eq_d);
                    PE_val_e = abs(y_e-y_eq_e)/(y_eq_e);
                    if(PE_val_a < tolerance && PE_val_b < tolerance && PE_val_c < tolerance && PE_val_d < tolerance && PE_val_e < tolerance) {
                        eq = 1;
                    } else {
                        eq = 0;
                    }

                    if(displayPEdata && !isnan(PE_val_a) && !isnan(PE_val_b) && !isnan(PE_val_c) && !isnan(PE_val_d) && !isnan(PE_val_e)) {
                        //only print if there are populations with which to calculate
                        printf("Reaction Group ID: %d, RG Class: %d\n",i, network.ReacGroups[i]);
                        for (int n = 0; n < 2; n++)
                            printf("Reactant[%d]: %d\n",n, network.reactant[n][i]);
                        for (int n = 0; n < 2; n++)
                            printf("Product[%d]: %d\n",n, network.product[n][i]);

                        printf("PE_val for y_a (Species: %d): %f\nPE_val for y_b (Species: %d): %f\nPE_val for y_c (Species: %d): %f\nPE_val for y_d (Species: %d): %f\nPE_val for y_e (Species: %d): %f\nIs this RG in equilibrium?: %d\n",
                            network.reactant[0][i],PE_val_a,
                            network.reactant[1][i],PE_val_b,
                            network.product[0][i],PE_val_c,
                            network.product[1][i],PE_val_d,
                            network.product[2][i],PE_val_e,
                            eq
                        );
                        printf("\n\n");
                    }
			
				}
				//reactions between 146 and 155 have carbon 12 and oxygen 16, and have non-zero starting populations
				//if (i > 146 && i < 155) 
				//if(displayPEdata)
					//printf("\nReactant/Product populations:\ny_a: %e\ny_b: %e\ny_c: %e\ny_d: %e\ny_e: %e\n\n", y_a, y_b, y_c, y_d, y_e);
			}
		}
	}

	/***End Partial Equilibrium***/

	/*
	   Begin the time integration from t=0 to tmax. Rather than t=0 we
	   start at some very small value of t. This is required for the CUDA C
	   code as well as the Java version.
	*/
	
	if (tid == 0)
	{
		t = 1.0e-20;
		dt = integrationData.dt_init;
		timesteps = 1;
	}
	
	fern_real floorFac = 0.1;
	fern_real upbumper = 0.9 * massTol;
	fern_real downbumper = 0.1;
	fern_real massTolUp = 0.25 * massTol;
	fern_real deltaTimeRestart = dt;
	fern_real dtFloor;
	fern_real dtFlux;
	fern_real massChecker;
	
	/* Compute mass numbers and initial mass fractions X for all isotopes. */
	
	for (int i = tid; i < numberSpecies; i += blockDim.x)
	{
		massNum[i] = (fern_real) Z[i] + (fern_real) N[i];
		/* Compute mass fraction X from abundance Y. */
		X[i] = massNum[i] * Y[i];
	}
	
	__syncthreads();
	sumXLast = NDreduceSum(X, numberSpecies);
	
	/* Main time integration loop */
	
	while (t < integrationData.t_max)
	{
		__syncthreads();
		/* Set Yzero[] to the values of Y[] updated in previous timestep. */
		
		for (int i = tid; i < numberSpecies; i += blockDim.x)
		{
			Yzero[i] = Y[i];
		}
		
		__syncthreads();
		
		/* Compute the fluxes from the previously-computed rates and the current abundances. */
		
		/* Parallel version of flux calculation */
		
		for (int i = tid; i < numberReactions; i += blockDim.x)
		{
			int nr = network.numReactingSpecies[i];
			Flux[i] = Rate[i] * Y[network.reactant[0][i]];
			
			switch (nr)
			{
			case 3:
				/* 3-body; flux = rate x Y x Y x Y */
				Flux[i] *= Y[network.reactant[2][i]];
				
			case 2:
				/* 2-body; flux = rate x Y x Y */
				Flux[i] *= Y[network.reactant[1][i]];
				break;
			}
		}
		
		__syncthreads();
		
		/* Populate the F+ and F- arrays in parallel from the master Flux array. */
		
		populateF(Fplus, FplusFac, Flux, MapFplus, totalFplus);
		populateF(Fminus, FminusFac, Flux, MapFminus, totalFminus);

		__syncthreads();

		/*
		   Sum the F+ and F- for each isotope. These are "sub-arrays"
		   of Fplus and Fminus at (F[+ or -] + minny) of size FplusMax[i].
		   The first loop applies to sub-arrays with size < 40. The outer
		   loop (in i) is parallel, but the inner loops (in j) are serial.

		   Alpha particles, protons, and neutrons all have size much
		   greater than 40, and they are summed in the next for loop, which
		   uses the NDreduceSum.
		*/
		
		int minny;
		
		for (int i = tid; i < numberSpecies; i += blockDim.x)
		{
            minny = (i > 0) ? FplusMax[i - 1] + 1 : 0;
			if ((FplusMax[i] + 1) - minny < 40)
			{
				/* Serially sum secction of F+. */
				FplusSum[i] = 0.0;
				for (int j = minny; j <= FplusMax[i]; j++)
				{
					FplusSum[i] += Fplus[j];
				}

				/* Serially sum section of F-. */
            	minny = (i > 0) ? FminusMax[i - 1] + 1 : 0;
				FminusSum[i] = 0.0;
				for (int j = minny; j <= FminusMax[i]; j++)
				{
					FminusSum[i] += Fminus[j];
				}
			}
		}
		
		for (int i = 0; i < numberSpecies; i++)
		{
            minny = (i > 0) ? FplusMax[i - 1] + 1 : 0;
			if ((FplusMax[i] + 1) - minny >= 40)
			{
				FplusSum[i] = NDreduceSum(Fplus + minny, (FplusMax[i] + 1) - minny);

            	minny = (i > 0) ? FminusMax[i - 1] + 1 : 0;
				FminusSum[i] = NDreduceSum(Fminus + minny, (FminusMax[i] + 1) - minny);
			}
		}

		__syncthreads();
		
		/* Find the maximum value of |FplusSum-FminusSum| to use in setting timestep. */
		
		for (int i = tid; i < numberSpecies; i += blockDim.x)
		{
			#ifdef FERN_SINGLE
				Fdiff[i] = fabsf(FplusSum[i] - FminusSum[i]);
			#else
				Fdiff[i] = fabs(FplusSum[i] - FminusSum[i]);
			#endif
		}
		
		__syncthreads();
		
		/* Call tree algorithm to find max of array Fdiff. */

		maxFlux = reduceMax(Fdiff, numberSpecies);

		__syncthreads();
		
		/*
		   Now use the fluxes to update the populations in parallel for this timestep.
		   For now we shall assume the asymptotic method. We determine whether each isotope
		   satisfies the asymptotic condition. If it does we update with the asymptotic formula.
		   If not, we update numerically using the forward Euler formula.
		*/
		
		/* Determine an initial trial timestep based on fluxes and dt in previous step. */
		
		if (tid == 0)
		{
			dtFlux = fluxFrac / maxFlux;
			dtFloor = floorFac * t;
			if (dtFlux > dtFloor) dtFlux = dtFloor;
			
			dt = dtFlux;
			if (deltaTimeRestart < dtFlux) dt = deltaTimeRestart;
		}
		
		__syncthreads();
		updatePopulations(FplusSum, FminusSum, Y, Yzero, numberSpecies, dt);
		__syncthreads();
		
		/* Compute sum of mass fractions sumX for all species. */
		
		for (int i = tid; i < numberSpecies; i += blockDim.x)
		{
			/* Compute mass fraction X from abundance Y. */
			X[i] = massNum[i] * Y[i];
		}
		
		__syncthreads();
		sumX = NDreduceSum(X, numberSpecies);
		
		__syncthreads();
		
		/*
		   Now modify timestep if necessary to ensure that particle number is conserved to
		   specified tolerance (but not too high a tolerance). Using updated populations
		   based on the trial timestep computed above, test for conservation of particle
		   number and modify trial timestep accordingly.
		*/
		
		if (tid == 0)
		{
			#ifdef FERN_SINGLE
				fern_real test1 = fabsf(sumXLast - 1.0);
				fern_real test2 = fabsf(sumX - 1.0);
				massChecker = fabsf(sumXLast - sumX);
						   
				if (test2 > test1 && massChecker > massTol)
				{
					dt *= fmaxf(massTol / fmaxf(massChecker, (fern_real) 1.0e-16), downbumper);
				}
				else if (massChecker < massTolUp)
				{
					dt *= (massTol / (fmaxf(massChecker, upbumper)));
				}
			#else
				fern_real test1 = fabs(sumXLast - 1.0);
				fern_real test2 = fabs(sumX - 1.0);
				massChecker = fabs(sumXLast - sumX);
						   
				if (test2 > test1 && massChecker > massTol)
				{
					dt *= fmax(massTol / fmax(massChecker, (fern_real) 1.0e-16), downbumper);
				}
				else if (massChecker < massTolUp)
				{
					dt *= (massTol / (fmax(massChecker, upbumper)));
				}

			#endif
		}
		
		__syncthreads();
		
		updatePopulations(FplusSum, FminusSum, Y, Yzero, numberSpecies, dt);
		
		__syncthreads();
		
		/*
		   Store the actual timestep that would be taken. Same as dt unless
		   artificially shortened in the last integration step to match end time.
		*/
		
		deltaTimeRestart = dt;
		
		/*
		   Finally check to be sure that timestep will not overstep next plot output
		   time and adjust to match if necessary. This will adjust dt only if at the end
		   of the integration interval. In that case it will also recompute the Y[]
		   corresponding to the adjusted time interval.
		 */
		
		if (t + dt >= integrationData.t_max)
		{
			if (tid == 0)
			{
				/*
				   TODO
				   Copy back to CPU for dt_init next operator split integration.
				   Params2[2] = dt;
				*/
				
				dt = integrationData.t_max - t;
			}
			
			__syncthreads();
			
			updatePopulations(FplusSum, FminusSum, Y, Yzero, numberSpecies, dt);
		}
		
		__syncthreads();
		
		/* NOTE: eventually need to deal with special case Be8 <-> 2 He4. */
		
		/* Now that final dt is set, compute final sum of mass fractions sumX. */
		
		for (int i = tid; i < numberSpecies; i += blockDim.x)
		{
			/* Compute mass fraction X from abundance Y. */
			X[i] = massNum[i] * Y[i];
		}
		
		__syncthreads();
		sumX = NDreduceSum(X, numberSpecies);
		
		__syncthreads();
		
		if (tid == 0)
		{
			/* Increment the integration time and set the new timestep. */
			
			t += dt;
			timesteps++;
		}
		
		sumXLast = sumX;
	}
}


/* Device functions */

/*
   Determines whether an isotope specified by speciesIndex satisfies the
   asymptotic condition. Returns 1 if it does and 0 if not.
*/

__device__ inline bool checkAsy(fern_real Fminus, fern_real Y, fern_real dt)
{
	/* This is not needed because 1.0 / 0.0 == inf in C and inf > 1.0 */
	
	/*
	   Prevent division by zero in next step
	   if (Y == 0.0)
	     return false;
	*/
	
	return (Fminus * dt / Y > 1.0);
}


/* Returns the updated Y using the asymptotic formula */

__device__ inline fern_real asymptoticUpdate(fern_real Fplus, fern_real Fminus, fern_real Y, fern_real dt)
{
	/* Sophia He formula */
	return (Y + Fplus * dt) / (1.0 + Fminus * dt / Y);
}


/* Returns the Y specified by speciesIndex updated using the forward Euler method */

__device__ inline fern_real eulerUpdate(fern_real FplusSum, fern_real FminusSum, fern_real Y, fern_real dt)
{
	return Y + (FplusSum - FminusSum) * dt;
}

/*
   Performs a parallel sum reduction in O(log(length)) time

   The given array is overwritten by intermediate values during computation.
   The maximum array size is 2 * blockDim.x.
*/

__device__ fern_real reduceSum(fern_real *a, unsigned short length)
{
	const int tid = threadIdx.x;
	unsigned short k = length;
	
	do
	{
		k = (k + 1) / 2;
		
		if (tid < k && tid + k < length)
			a[tid] += a[tid + k];
		
		length = k;
		__syncthreads();
	}
	while (k > 1);
	
	return a[0];
}

/*
   Non-destructive sum reduction.
   Same as previous, but copies array to dsmem allocated
   to the global scratch_space before executing algorithm.
*/

__device__ fern_real NDreduceSum(fern_real *a, unsigned short length)
{
    const int tid = threadIdx.x;
    unsigned short k = length;
    fern_real *b;

    b = scratch_space;

    for (int i = tid; i < k; i += blockDim.x) {
        b[i] = a[i];
    }

    __syncthreads();

    do {
        k = (k + 1) / 2;

        if (tid < k && tid + k < length)
            b[tid] += b[tid + k];

        length = k;
        __syncthreads();
    } while (k > 1);

    return b[0];
}

/*
   Performs a parallel maximum reduction in O(log(length)) time

   The given array is overwritten by intermediate values during computation.
   The maximum array size is 2 * blockDim.x.
*/
__device__ fern_real reduceMax(fern_real *a, unsigned short length)
{
	const int tid = threadIdx.x;
	unsigned short k = length;
	
	do
	{
		k = (k + 1) / 2;
		
		if (tid < k && tid + k < length)
			#ifdef FERN_SINGLE
				a[tid] = fmaxf(a[tid], a[tid + k]);
			#else
				a[tid] = fmax(a[tid], a[tid + k]);
			#endif
		
		length = k;
		__syncthreads();
	}
	while (k > 1);
	
	return a[0];
}

/*
   Populates Fplus or Fminus
   Since the calculations for Fplus and Fminus are similar, the implementation
   of this function uses the term 'sign' to replace 'plus' and 'minus'.
*/

__device__ void populateF(fern_real *Fsign, fern_real *FsignFac, fern_real *Flux,
	unsigned short *MapFsign, unsigned short totalFsign)
{
	const int tid = threadIdx.x;
	
	for (int i = tid; i < totalFsign; i += blockDim.x)
	{
		Fsign[i] = FsignFac[i] * Flux[MapFsign[i]];
	}
}


/* Updates populations based on the trial timestep */

__device__ inline void updatePopulations(fern_real *FplusSum, fern_real *FminusSum,
	fern_real *Y, fern_real *Yzero, unsigned short numberSpecies, fern_real dt)
{
	const int tid = threadIdx.x;
	
	/* Parallel Update populations based on this trial timestep. */
	for (int i = tid; i < numberSpecies; i += blockDim.x)
	{
		if (checkAsy(FminusSum[i], Y[i], dt))
		{
			Y[i] = asymptoticUpdate(FplusSum[i], FminusSum[i], Yzero[i], dt);
		}
		else
		{
			Y[i] = eulerUpdate(FplusSum[i], FminusSum[i], Yzero[i], dt);
		}
	}
}

__device__ void network_print(const Network &network)
{
	/* Network data */
	
	printf("species: %d\n", network.species);
	
	printf("Z: { ");
	for (int i = 0; i < network.species; i++)
		printf("%4d ", network.Z[i]);
	printf("}\n");
	
	printf("N: { ");
	for (int i = 0; i < network.species; i++)
		printf("%4d ", network.N[i]);
	printf("}\n");
	
	/* Reaction data */
	
	printf("\n");
	
	printf("reactions: %d\n", network.reactions);
	
	for (int n = 0; n < 7; n++)
	{
		printf("P[%d]: { ", n);
		for (int i = 0; i < network.reactions; i++)
			printf("%e ", network.P[n][i]);;
		printf("\n");
	}
	
	printf("numReactingSpecies: { ");
	for (int i = 0; i < network.reactions; i++)
		printf("%4d ", network.numReactingSpecies[i]);
	printf("}\n");
	
	
	printf("statFac: { ");
	for (int i = 0; i < network.reactions; i++)
		printf("%e ", network.statFac[i]);
	printf("}\n");
	
	
	printf("Q: { ");
	for (int i = 0; i < network.reactions; i++)
		printf("%e ", network.Q[i]);
	printf("}\n");
	
	for (int n = 0; n < 3; n++)
	{
		printf("reactant[%d]: { ", n);
		for (int i = 0; i < network.reactions; i++)
			printf("%4d ", network.reactant[n][i]);
		printf("}\n");
	}
	
	printf("totalFplus: %d\n", network.totalFplus);
	printf("totalFminus: %d\n", network.totalFminus);
	
	printf("FplusFac: { ");
	for (int i = 0; i < network.totalFplus; i++)
		printf("%e ", network.FplusFac[i]);
	printf("}\n");
	
	printf("FminusFac: { ");
	for (int i = 0; i < network.totalFminus; i++)
		printf("%e ", network.FminusFac[i]);
	printf("}\n");
	
	printf("MapFplus: { ");
	for (int i = 0; i < network.totalFplus; i++)
		printf("%4u ", network.MapFplus[i]);
	printf("}\n");
	
	printf("MapFminus: { ");
	for (int i = 0; i < network.totalFminus; i++)
		printf("%4u ", network.MapFminus[i]);
	printf("}\n");
	
	printf("FplusMax: { ");
	for (int i = 0; i < network.species; i++)
		printf("%4u ", network.FplusMax[i]);
	printf("}\n");
	
	printf("FminusMax: { ");
	for (int i = 0; i < network.species; i++)
		printf("%4u ", network.FminusMax[i]);
	printf("}\n");
}

size_t integrateNetwork_sharedSize(const Network &network)
{
	size_t size = 49100;

	return size;
}
