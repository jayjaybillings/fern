
#include <stdlib.h>
#include <stdio.h>
#include "FERNIntegrator.cuh"

int main(int argc, char const *argv[])
{
	/* Load the network */
	FERNIntegrator integrator;
	integrator.network.species = 150;
	integrator.network.reactions = 1604;
	integrator.network.massTol = 1.0e-7;
	integrator.network.fluxFrac = 0.01;

	integrator.network.allocate();
	integrator.network.loadNetwork("CUDAnet_150.inp");
	integrator.network.loadReactions("rateLibrary_150.data");

	/* Initialize the solver */

	integrator.initializeCuda();
	integrator.prepareKernel();

	// Create the unique integration data

	{
		IntegrationData integrationData;
		integrationData.allocate(integrator.network.species);
		integrationData.loadAbundances("CUDAnet_150.inp");
		
		integrationData.T9 = 7.0;
		integrationData.t_init = 1.0e-20;
		integrationData.t_max = 1.0e-9;
		integrationData.dt_init = 1.23456789e-22;
		integrationData.rho = 1.0e8;

		// Launch the kernel
		
		integrator.integrate(integrationData);
		
		integrationData.print(&integrator.network);
	}
	
	return EXIT_SUCCESS;
}
