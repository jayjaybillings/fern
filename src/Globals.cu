#include "Globals.cuh"
#include "Network.cuh"


void Globals::cudaAllocate(const Network &network)
{
	hipMalloc(&preFac, sizeof(fern_real) * network.reactions);
	hipMalloc(&Fplus, sizeof(fern_real) * network.totalFplus);
	hipMalloc(&Fminus, sizeof(fern_real) * network.totalFminus);
	hipMalloc(&rate, sizeof(fern_real) * network.reactions);
	hipMalloc(&massNum, sizeof(fern_real) * network.species);
	hipMalloc(&X, sizeof(fern_real) * network.species);
	hipMalloc(&Fdiff, sizeof(fern_real) * network.species);
	hipMalloc(&Yzero, sizeof(fern_real) * network.species);
	hipMalloc(&FplusSum, sizeof(fern_real) * network.totalFplus);
	hipMalloc(&FminusSum, sizeof(fern_real) * network.totalFminus);
	hipMalloc(&Flux, sizeof(fern_real) * network.reactions);
}
